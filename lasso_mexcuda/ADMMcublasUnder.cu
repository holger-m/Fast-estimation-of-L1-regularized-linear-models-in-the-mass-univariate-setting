#include "hip/hip_runtime.h"

// compile: system('nvcc -c ADMMcublasUnder.cu');
        
// alternatively (on Windows): system('nvcc -c ADMMcublasUnder.cu -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx64\x64"');

#include "ADMMcublasUnder.h"
#include "math.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

__global__ void soft_thres(float *x_out, float *u_out, float *z_out, float const * const lambda_value_in, int N_n)
{
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N_n){
        
        z_out[i] = fmaxf(x_out[i] + u_out[i] - lambda_value_in[0], 0.0f) - fmaxf(-x_out[i] - u_out[i] - lambda_value_in[0], 0.0f);
        
    }

}

__global__ void delta_abs_value(float *delta_abs_out, int N_n)
{
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N_n){
        
        delta_abs_out[i] = fabsf(delta_abs_out[i]);
        
    }

}

__global__ void determine_convergence(float *delta_abs_out, int max_index, float *z_old_out, int max_index_dual, float const * const tol_value_in, bool *conv_bool_out)
{
    
    if (delta_abs_out[max_index-1] < tol_value_in[0] && z_old_out[max_index_dual-1] < tol_value_in[0]){  // 1-based index returned by hipblasIsamax
    
        conv_bool_out[0] = true;
        
    }
    else
    {
        
        conv_bool_out[0] = false;
        
    }

}

void ADMM_cublas_under(int N_j, int N_n, int N_batch, int n_iter_max, float *z_in_host, float *u_in_host, float *lambda_value_in_host, float *Atb_active_in_host, float *LU_inv_in_host, float *tol_value_in_host, float *z_host_out, float *u_host_out, bool *error_flag_max_iter)
{
    
    float *dz_out;
    hipMalloc(&dz_out, N_n*sizeof(float));

    float *du_out;
    hipMalloc(&du_out, N_n*sizeof(float));

    float *dlambda_value_in;
    hipMalloc(&dlambda_value_in, sizeof(float));
    
    float *dAtb_active_in;
    hipMalloc(&dAtb_active_in, N_n*sizeof(float));

    float *dLU_inv_in;
    hipMalloc(&dLU_inv_in, (N_j*N_j)*sizeof(float));

    float *dtol_value_in;
    hipMalloc(&dtol_value_in, sizeof(float));

    hipMemcpy(dz_out, z_in_host, N_n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(du_out, u_in_host, N_n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dlambda_value_in, lambda_value_in_host, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dAtb_active_in, Atb_active_in_host, N_n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dLU_inv_in, LU_inv_in_host, (N_j*N_j)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dtol_value_in, tol_value_in_host, sizeof(float), hipMemcpyHostToDevice);

    float *ddelta_abs_out;
    hipMalloc(&ddelta_abs_out, N_n*sizeof(float));

    float *dq_out;
    hipMalloc(&dq_out, N_n*sizeof(float));

    float *dx_out;
    hipMalloc(&dx_out, N_n*sizeof(float));

    float *dz_old_out;
    hipMalloc(&dz_old_out, N_n*sizeof(float));

    bool *dconv_bool_out;
    hipMalloc(&dconv_bool_out, sizeof(bool));  

    float scalar_p1;
    float scalar_0;
    float scalar_m1;
    int const threadsPerBlock = 256;
    int blocksPerGrid;
    int max_index;
    int max_index_dual;
    bool *conv_bool_host = new bool[1];
    int iter_no;

    scalar_p1 = 1.0f;
    scalar_0 = 0.0f;
    scalar_m1 = -1.0f;

    blocksPerGrid = (N_n + threadsPerBlock - 1) / threadsPerBlock;

    hipblasHandle_t handle;

    hipblasCreate(&handle); 
    
    for (iter_no = 0; iter_no < n_iter_max; iter_no++) {
        
        // ADMM
        
        hipblasScopy(handle, N_n, dz_out, 1, dq_out, 1);

        hipblasSaxpy(handle, N_n, &scalar_m1, du_out, 1, dq_out, 1);

        hipblasSaxpy(handle, N_n, &scalar_p1, dAtb_active_in, 1, dq_out, 1);

        hipblasSsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, N_j, N_batch, &scalar_p1, dLU_inv_in, N_j, dq_out, N_j, &scalar_0, dx_out, N_j);

        hipblasScopy(handle, N_n, dz_out, 1, dz_old_out, 1);

        soft_thres<<<blocksPerGrid, threadsPerBlock>>>(dx_out, du_out, dz_out, dlambda_value_in, N_n);
    
        hipblasSaxpy(handle, N_n, &scalar_p1, dx_out, 1, du_out, 1);

        hipblasSaxpy(handle, N_n, &scalar_m1, dz_out, 1, du_out, 1);

        // prim. conv

        hipblasScopy(handle, N_n, dx_out, 1, ddelta_abs_out, 1);

        hipblasSaxpy(handle, N_n, &scalar_m1, dz_out, 1, ddelta_abs_out, 1);

        delta_abs_value<<<blocksPerGrid, threadsPerBlock>>>(ddelta_abs_out, N_n);

        hipblasIsamax(handle, N_n, ddelta_abs_out, 1, &max_index);

        // dual conv
        
        hipblasSaxpy(handle, N_n, &scalar_m1, dz_out, 1, dz_old_out, 1);

        delta_abs_value<<<blocksPerGrid, threadsPerBlock>>>(dz_old_out, N_n);

        hipblasIsamax(handle, N_n, dz_old_out, 1, &max_index_dual);
        
        // both conv
        
        determine_convergence<<<1,1>>>(ddelta_abs_out, max_index, dz_old_out, max_index_dual, dtol_value_in, dconv_bool_out);

        hipMemcpy(conv_bool_host, dconv_bool_out, sizeof(bool), hipMemcpyDeviceToHost);

        if (conv_bool_host[0])
        {
            
            break;
            
        }

        if (iter_no == (n_iter_max - 1)){

            error_flag_max_iter[0] = true;

        }
        
    }

    hipMemcpy(z_host_out, dz_out, N_n*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(u_host_out, du_out, N_n*sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(dz_out);
    hipFree(du_out);
    hipFree(ddelta_abs_out);
    hipFree(dq_out);
    hipFree(dx_out);
    hipFree(dz_old_out);
    hipFree(dconv_bool_out);

    hipFree(dlambda_value_in);
    hipFree(dAtb_active_in);
    hipFree(dLU_inv_in);
    hipFree(dtol_value_in);
    
}
